
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

#define CEIL(a, b) ((a-1)/b +1)

const int MAX_DIM = 100;
const int MAX_SIZE = MAX_DIM*MAX_DIM;
const int MAX_BYTES = MAX_SIZE*sizeof(float);

__global__ void matrix_mul(float *d_m1, float *d_m2, float *d_m3){
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    if(idx >= MAX_SIZE)
        return;
    float tempsum=0;
    int i = idx/MAX_DIM;
    int j = idx%MAX_DIM;
    for(int k=0;k<MAX_DIM;k++){
        tempsum += d_m1[i*MAX_DIM + k]*d_m2[j + k*MAX_DIM];
    }
    d_m3[idx] = tempsum;
}

int main(){
printf("starting program execution\n");fflush(stdin);
    //allocating memory for host arrays
    float h_m1[MAX_SIZE], h_m2[MAX_SIZE], h_m3[MAX_SIZE];
printf("host arrays declared\n");fflush(stdin);
    //generating input arrays
    for(int i=0;i<MAX_SIZE;i++)
        h_m1[i] = (float)(rand()%100);
    for(int i=0;i<MAX_SIZE;i++)
        h_m2[i] = (float)(rand()%100);
printf("host arrays generated\n");fflush(stdin);   
    //declaring device memory pointers
    float *d_m1, *d_m2, *d_m3;
printf("device pointers declared\n");fflush(stdin);
    //allocating device memory
    hipMalloc((void **)&d_m1, MAX_BYTES);
    hipMalloc((void **)&d_m2, MAX_BYTES);
    hipMalloc((void **)&d_m3, MAX_BYTES);
printf("device memory allocated\n");fflush(stdin);
    //copying data from host to device
    hipMemcpy(d_m1, h_m1, MAX_BYTES, hipMemcpyHostToDevice);
    hipMemcpy(d_m2, h_m2, MAX_BYTES, hipMemcpyHostToDevice);
printf("starting kernel call\n");fflush(stdin);
    //calling kernel
    matrix_mul<<< CEIL(MAX_SIZE, 1024), 1024 >>>(d_m1, d_m2, d_m3);
printf("kernel call complete\n");fflush(stdin);
    //transferring result from device to host
    hipMemcpy(h_m3, d_m3, MAX_BYTES, hipMemcpyDeviceToHost);

    //checking correctness of answer
    int flag =1;
    for(int i=0;i<MAX_DIM;i++){
        for(int j=0;j<MAX_DIM;j++){
            float tempsum=0;
            for(int k=0;k<MAX_DIM;k++){
                tempsum += h_m1[i*MAX_DIM + k] * h_m2[j + k*MAX_DIM];
            }
            if(h_m3[i*MAX_DIM+j] != tempsum){
                printf("wrong value at %d\n",i*MAX_DIM+j);
                printf("Expected value:%f, found value:%f\n",tempsum, h_m3[i*MAX_DIM+j]);
                flag=0;
                break;
            }
        }
        if(flag==0)
            break;
    }
    if(flag==1)
        printf("The solution is correct\n");
}